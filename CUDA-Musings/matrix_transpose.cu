#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../../common/book.h"

const int ROWS = 3000;
const int COLS = 2000;


// Arguments are pointers to 
__global__ void transpose(int (*a)[COLS], int (*t)[ROWS]) {
  int cur_row = blockIdx.y;
  int cur_col = blockIdx.x;
  t[cur_row][cur_col] = a[cur_col][cur_row];
}


int main( void ) {
 
  // Declare arrays to be used on host (allocate memory off heap to prevent segfault)
  int *A = (int *)malloc(ROWS * COLS * sizeof(int));
  int *T = (int *)malloc(ROWS * COLS * sizeof(int));

  // Declare pointers to be used/evaluated on device
  // Pointers must be of type array containing number of elements in first element of 2D array
  int (*Aptr)[COLS];
  int (*Tptr)[ROWS];

  dim3 grid(ROWS,COLS);

  // Allocate space for the pointers on the device
  HANDLE_ERROR( hipMalloc( (void**)&Aptr, ROWS * COLS * sizeof(int) ) ); 
  HANDLE_ERROR( hipMalloc( (void**)&Tptr, ROWS * COLS * sizeof(int) ) );

  // Fill up matrix A
  printf("Filling A matrix...\n");
  int fill = 0;
  for(int j = 0; j < ROWS; j++) {
    for(int i = 0; i < COLS; i++) {
      A[i+j*COLS] = fill;
      fill++;
      //printf("A[%d][%d] = %d\n",j,i,A[i+j*COLS]);
	}
  }

  // Point data stored in A on host to device with memory address: Aptr
  HANDLE_ERROR( hipMemcpy( Aptr, A, ROWS * COLS * sizeof(int), hipMemcpyHostToDevice ) );

  printf("Transposing matrix...\n");
  transpose<<<grid,1>>>(Aptr, Tptr); // Two-dimensional grid of blocks simulates matrix

  // Copy back transposed matrix (Tptr) to host
  HANDLE_ERROR( hipMemcpy( T, Tptr, ROWS * COLS * sizeof(int), hipMemcpyDeviceToHost ) );

  // Check to see if transposition worked
  int success = 1;
  for(int j = 0; j < COLS; j++) {
    for(int i = 0; i < ROWS; i++) {      
      //printf("T[%d][%d] = %d\n",j,i,T[i+j*ROWS]);
      if (T[i+j*ROWS] != A[j+i*COLS])
      	 success = 0;
    }
  }

  if (success) {
     printf("Success!\n");
  }

  else {
     printf("Failure\n");
  }

 
  hipFree( Aptr );
  hipFree( Tptr );
  free( A );
  free( T );

  return 0;
}
